
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

extern "C"
{
    __device__ float point_to_point(hipFloatComplex p1, hipFloatComplex p2)
    {
        float dx = hipCrealf(p1) - hipCrealf(p2);
        float dy = hipCimagf(p1) - hipCimagf(p2);
        return sqrtf(dx * dx + dy * dy);
    }

    __device__ float point_to_segment(hipFloatComplex p, hipFloatComplex seg_a, hipFloatComplex seg_b)
    {
        hipFloatComplex ab = make_hipFloatComplex(hipCrealf(seg_b) - hipCrealf(seg_a), hipCimagf(seg_b) - hipCimagf(seg_a));
        hipFloatComplex ap = make_hipFloatComplex(hipCrealf(p) - hipCrealf(seg_a), hipCimagf(p) - hipCimagf(seg_a));
        float proj_coeff = fmaxf(0, fminf(1, (hipCrealf(ab) * hipCrealf(ap) + hipCimagf(ab) * hipCimagf(ap)) / (hipCrealf(ab) * hipCrealf(ab) + hipCimagf(ab) * hipCimagf(ab))));
        hipFloatComplex closest_point = make_hipFloatComplex(hipCrealf(seg_a) + proj_coeff * hipCrealf(ab), hipCimagf(seg_a) + proj_coeff * hipCimagf(ab));
        return point_to_point(closest_point, p);
    }

    __global__ void spd_cross_product(const hipFloatComplex *target_trajectories, int num_targets,
                                      const hipFloatComplex *trajectories, int num_trajectories,
                                      int traj_len, float *results)
    {
        // Calculate the indices for the current thread
        int traj_idx = blockIdx.x;   // Index of the trajectory we're processing
        int target_idx = blockIdx.y; // Index of the target trajectory we're comparing against
        int point_idx = threadIdx.x; // Index of the point within the trajectory

        // Check if the current thread is within bounds
        if (traj_idx < num_trajectories && target_idx < num_targets && point_idx < traj_len)
        {
            // Get the current point from the trajectory we're processing
            hipFloatComplex p = trajectories[traj_idx * traj_len + point_idx];

            // Initialize the minimum distance to a very large value (similar to FLT_MAX)
            float min_dist = 3.402823466e+38f;

            // Iterate through each segment of the target trajectory
            for (int i = 0; i < traj_len - 1; ++i)
            {
                // Get the start and end points of the current segment in the target trajectory
                hipFloatComplex a = target_trajectories[target_idx * traj_len + i];
                hipFloatComplex b = target_trajectories[target_idx * traj_len + i + 1];

                // Calculate the distance from the current point to the current segment
                float dist = point_to_segment(p, a, b);

                // Update the minimum distance if we found a closer segment
                if (dist < min_dist)
                {
                    min_dist = dist;
                }
            }

            // Add the minimum distance (divided by trajectory length) to the result
            // We use atomicAdd because multiple threads might be updating the same result
            atomicAdd(&results[traj_idx * num_targets + target_idx], min_dist / traj_len);
        }
    }

    __global__ void spd_pairwise(const hipFloatComplex *target_trajectories,
                                 const hipFloatComplex *trajectories,
                                 int num_trajectories, int traj_len, float *results)
    {
        // Calculate the indices for the current thread
        int traj_idx = blockIdx.x;   // Index of the trajectory pair we're processing
        int point_idx = threadIdx.x; // Index of the point within the trajectory

        // Check if the current thread is within bounds
        if (traj_idx < num_trajectories && point_idx < traj_len)
        {
            // Get the current point from the trajectory we're processing
            hipFloatComplex p = trajectories[traj_idx * traj_len + point_idx];

            // Initialize the minimum distance to a very large value (similar to FLT_MAX)
            float min_dist = 3.402823466e+38f;

            // Iterate through each segment of the corresponding target trajectory
            for (int i = 0; i < traj_len - 1; ++i)
            {
                // Get the start and end points of the current segment in the target trajectory
                hipFloatComplex a = target_trajectories[traj_idx * traj_len + i];
                hipFloatComplex b = target_trajectories[traj_idx * traj_len + i + 1];

                // Calculate the distance from the current point to the current segment
                float dist = point_to_segment(p, a, b);

                // Update the minimum distance if we found a closer segment
                if (dist < min_dist)
                {
                    min_dist = dist;
                }
            }

            // Add the minimum distance (divided by trajectory length) to the result
            // We use atomicAdd because multiple threads might be updating the same result
            atomicAdd(&results[traj_idx], min_dist / traj_len);
        }
    }
}