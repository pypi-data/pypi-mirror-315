#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <float.h>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/random.h>

__device__ float point_to_point(hipFloatComplex p1, hipFloatComplex p2)
{
    float dx = hipCrealf(p1) - hipCrealf(p2);
    float dy = hipCimagf(p1) - hipCimagf(p2);
    return sqrtf(dx * dx + dy * dy);
}

__device__ float point_to_segment(hipFloatComplex p, hipFloatComplex seg_a, hipFloatComplex seg_b)
{
    // vactor ab
    hipFloatComplex ab = make_hipFloatComplex(hipCrealf(seg_b) - hipCrealf(seg_a), hipCimagf(seg_b) - hipCimagf(seg_a));
    // vector ap
    hipFloatComplex ap = make_hipFloatComplex(hipCrealf(p) - hipCrealf(seg_a), hipCimagf(p) - hipCimagf(seg_a));
    // projection of `p` onto `ab`
    // coeff = (abx*apx + aby*apy) / (abx*abx + aby*aby)
    // dx = ax + abx * coeff
    // dy = ay + aby * coeff
    float proj_coeff = fmaxf(0, fminf(1, (hipCrealf(ab) * hipCrealf(ap) + hipCimagf(ab) * hipCimagf(ap)) / (hipCrealf(ab) * hipCrealf(ab) + hipCimagf(ab) * hipCimagf(ab))));
    // if coeff is 1 ==> the distance between p and seg is the distance between p and b
    // if coef is 0 ==> the distance between p and seg is the distance between p and a
    // if coef is in (0, 1) the distance between p and seg is the distance between p and its projection onto ab
    hipFloatComplex closest_dist = make_hipFloatComplex(hipCrealf(seg_a) + proj_coeff * hipCrealf(ab), hipCimagf(seg_a) + proj_coeff * hipCimagf(ab));
    return point_to_point(closest_dist, p);
}

struct Point_To_Trajectory_Functor
{
    hipFloatComplex *segments;
    int len;

    Point_To_Trajectory_Functor(hipFloatComplex *s, int l) : segments(s), len(l) {}

    __device__ float operator()(hipFloatComplex p) const
    {
        float min_dist = FLT_MAX;
        for (int i = 0; i < len; ++i)
        {
            hipFloatComplex a = segments[2 * i];
            hipFloatComplex b = segments[2 * i + 1];
            float dist = point_to_segment(p, a, b);
            if (dist < min_dist)
            {
                min_dist = dist;
            }
        }
        return min_dist;
    }
};

float D_spd(thrust::device_vector<hipFloatComplex> &points, thrust::device_vector<hipFloatComplex> &segments)
{
    int len_points = points.size();
    int len_segments = segments.size() / 2;

    // Apply the Point_To_Trajectory_Functor to each point in parallel
    float total_distance = thrust::transform_reduce(
        points.begin(), points.end(),
        Point_To_Trajectory_Functor(thrust::raw_pointer_cast(segments.data()), len_segments),
        0.0f, thrust::plus<float>());

    // Calculate the mean distance
    return total_distance / len_points;
}

float compute_sspd(std::vector<hipFloatComplex> &segments1, std::vector<hipFloatComplex> &segments2)
{
    thrust::device_vector<hipFloatComplex> d_segments1(segments1);
    thrust::device_vector<hipFloatComplex> d_segments2(segments2);

    float d_spd_1_to_2 = D_spd(d_segments1, d_segments2);
    float d_spd_2_to_1 = D_spd(d_segments2, d_segments1);

    return (d_spd_1_to_2 + d_spd_2_to_1) / 2.0f;
}

int main()
{
    // Example usage
    std::vector<hipFloatComplex> segments1 = {
        make_hipFloatComplex(0.0f, 0.0f), make_hipFloatComplex(1.0f, 1.0f),
        make_hipFloatComplex(1.0f, 1.0f), make_hipFloatComplex(2.0f, 2.0f)};

    std::vector<hipFloatComplex> segments2 = {
        make_hipFloatComplex(2.0f, 0.0f), make_hipFloatComplex(3.0f, 1.0f),
        make_hipFloatComplex(3.0f, 1.0f), make_hipFloatComplex(4.0f, 2.0f)};

    float sspd = compute_sspd(segments1, segments2);
    std::cout << "SSPD: " << sspd << std::endl;

    return 0;
}
